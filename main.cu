#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define NUM_THREADS 1024
#define NUM_BLOCKS 80
#define FLOP_COUNT 10000

#define OPERATION for (int j = 0; j < FLOP_COUNT; ++j){ \
                    d_data[tid] *= 2.0f;                \
                  }                                        

/********************************

    Device Functions/Functors

********************************/

class test_runner { 

    private:
        int x = 1;

    public: 
        __device__ __noinline__ void operator()(float *d_data) {
        int tid = threadIdx.x;
        if (threadIdx.x == 0){
            atomicAdd(&x, 1);
        }

        OPERATION
    }

};

//constant functor versions
__constant__ test_runner object;
__constant__ test_runner* object_ptr = &object;

//non-constant functor versions
__device__ test_runner object_dev;

//functions that will either be inlined or not inlined
__device__ __noinline__ void perform_calculations(float *d_data) {
    int tid = threadIdx.x;

    OPERATION
}

__device__ void perform_calculations_inline(float *d_data) {
    int tid = threadIdx.x;

    OPERATION
}

/********************************

           Kernels

********************************/

// Kernel launches the __device__ function on each thread
__global__ void scenario1(float *d_data) {
    perform_calculations(d_data);
}

// Function for the single thread that launches a dynamic block
__global__ void scenario2(float *d_data, int depth = 0) {
    int tid = threadIdx.x;

    if (depth == 1){

        OPERATION

        return;
    }

    if (tid == 0) {

        // dynamic parallelism block with 1024 threads
        float *d_temp;
        hipMalloc(&d_temp, NUM_THREADS * sizeof(float));
        scenario2<<<NUM_BLOCKS, NUM_THREADS>>>(d_temp, 1);
        hipDeviceSynchronize();
    } 

    __syncthreads();
}

// Kernel launches the inlined __device__ function on each thread
__global__ void scenario3(float *d_data) {
    perform_calculations_inline(d_data);
}

// Kernel launches the __constant__ functor on each thread
__global__ void scenario4(float *d_data) {

    object(d_data);
}

// Kernel launches the __device__ functor on each thread
__global__ void scenario5(float *d_data) {

    object_dev(d_data);
}

int main() {

    //vars
    float *h_data, *d_data1, *d_data2, *d_data3, *d_data4, *d_data5;

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    float scenario1_time_ms, scenario2_time_ms, scenario3_time_ms, scenario4_time_ms, scenario5_time_ms;

    //Make memory for random task usage
    //(replace with what you want to test)
    h_data = new float[NUM_THREADS];
    hipMalloc(&d_data1, NUM_THREADS * sizeof(float));
    hipMalloc(&d_data2, NUM_THREADS * sizeof(float));
    hipMalloc(&d_data3, NUM_THREADS * sizeof(float));
    hipMalloc(&d_data4, NUM_THREADS * sizeof(float));
    hipMalloc(&d_data5, NUM_THREADS * sizeof(float));

    for (int i = 0; i < NUM_THREADS; ++i) h_data[i] = 1.0f;

    hipMemcpy(d_data1, h_data, NUM_THREADS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data2, h_data, NUM_THREADS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data3, h_data, NUM_THREADS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data4, h_data, NUM_THREADS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data5, h_data, NUM_THREADS * sizeof(float), hipMemcpyHostToDevice);

    //Warm the device up to account for potential overhead
    scenario1<<<NUM_BLOCKS, NUM_THREADS>>>(d_data1);
    scenario2<<<NUM_BLOCKS, NUM_THREADS>>>(d_data2);
    scenario3<<<NUM_BLOCKS, NUM_THREADS>>>(d_data3);
    scenario3<<<NUM_BLOCKS, NUM_THREADS>>>(d_data4);
    scenario3<<<NUM_BLOCKS, NUM_THREADS>>>(d_data5);
    hipDeviceSynchronize();

    //Run Scenario 1
    hipEventRecord(start_event, 0);
    scenario1<<<NUM_BLOCKS, NUM_THREADS>>>(d_data1);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);

    hipEventElapsedTime(&scenario1_time_ms, start_event, stop_event);

    //Run Scenario 2
    hipEventRecord(start_event, 0);
    scenario2<<<NUM_BLOCKS, NUM_THREADS>>>(d_data2);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);

    hipEventElapsedTime(&scenario2_time_ms, start_event, stop_event);

    //Run Scenario 3
    hipEventRecord(start_event, 0);
    scenario3<<<NUM_BLOCKS, NUM_THREADS>>>(d_data3);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);

    hipEventElapsedTime(&scenario3_time_ms, start_event, stop_event);

    //Run Scenario 4
    hipEventRecord(start_event, 0);
    scenario4<<<NUM_BLOCKS, NUM_THREADS>>>(d_data4);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);

    hipEventElapsedTime(&scenario4_time_ms, start_event, stop_event);

    //Run Scenario 5
    hipEventRecord(start_event, 0);
    scenario5<<<NUM_BLOCKS, NUM_THREADS>>>(d_data5);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);

    hipEventElapsedTime(&scenario5_time_ms, start_event, stop_event);

    //Print times
    std::cout << "Scenario: (1024 threads, inlined kernels) time: " << scenario3_time_ms << " ms\n";
    std::cout << "Scenario: (1024 threads, individual kernels) time: " << scenario1_time_ms << " ms\n";
    std::cout << "Scenario: (1024 threads, dynamic parallelism) time: " << scenario2_time_ms << " ms\n";
    std::cout << "Scenario: (1024 threads, polymorphic functor mimic [device]) time: " << scenario5_time_ms << " ms\n";
    std::cout << "Scenario: (1024 threads, polymorphic functor mimic [constant]) time: " << scenario4_time_ms << " ms\n";

    return 0;
}
